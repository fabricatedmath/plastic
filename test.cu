#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <Eigen/Dense>
#include "test.h"

using namespace std;
using namespace Eigen;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct CudaMatrixXf {
    float* data;
    size_t pitch;
};

struct CudaVectorXf {
    float* data;
};

struct CudaTest {
    CudaMatrixXf m;
    CudaVectorXf v;
};

__device__ float* getRowPtr(CudaMatrixXf,int);

__global__ void test_kernel(CudaTest test, unsigned long long* time) {
    unsigned long long startTime = clock();
    float v = test.v.data[threadIdx.x];
    test.v.data[threadIdx.x] = v+1;
    for (int row = 0; row < 5; row++) {
        float* rowPtr = getRowPtr(test.m, row);
        float i = rowPtr[threadIdx.x];
//        printf("%d:%f:%d\n",row, i, threadIdx.x);
        rowPtr[threadIdx.x] = i+1;
    }
    unsigned long long endTime = clock();
    *time = (endTime - startTime);
}

hipError_t hipMalloc(VectorXf* v, CudaVectorXf* cv) {
    return hipMalloc((void**)&cv->data, v->size() * sizeof(float));
}

hipError_t memcpyHostToDevice(VectorXf* v, CudaVectorXf* cv) {
    return hipMemcpy((void**)cv->data, v->data(), v->size() * sizeof(float), hipMemcpyHostToDevice);
}

hipError_t memcpyDeviceToHost(VectorXf* v, CudaVectorXf* cv) {
    return hipMemcpy((void**)v->data(), cv->data, v->size() * sizeof(float), hipMemcpyDeviceToHost);
}

hipError_t hipMalloc(MatrixXf* m, CudaMatrixXf* cm) {
   return hipMallocPitch((void**)&cm->data, &cm->pitch, m->cols() * sizeof(float), m->rows());
}

hipError_t memcpyHostToDevice(MatrixXf* m, CudaMatrixXf* cm) {
    return hipMemcpy2D(cm->data, cm->pitch, m->data(), m->cols() * sizeof(float), m->cols() * sizeof(float), m->rows(), hipMemcpyHostToDevice);
}

hipError_t memcpyDeviceToHost(MatrixXf* m, CudaMatrixXf* cm) {
    return hipMemcpy2D(m->data(), m->cols() * sizeof(float), cm->data, cm->pitch, m->cols() * sizeof(float), m->rows(), hipMemcpyDeviceToHost);
}

float* getRowPtr(CudaMatrixXf cm, int row) {
    return (float*)((char*)cm.data + row*cm.pitch);
}

void wrapper(Test test) {
    unsigned long long time;
    unsigned long long* d_time;
    gpuErrchk( hipMalloc(&d_time, sizeof(unsigned long long)) );
    CudaTest cudaTest;
    gpuErrchk( hipMalloc(&test.v, &cudaTest.v) );
    gpuErrchk( memcpyHostToDevice(&test.v, &cudaTest.v) );
    gpuErrchk( hipMalloc(&test.m, &cudaTest.m) );
    gpuErrchk( memcpyHostToDevice(&test.m, &cudaTest.m) );
    for (int i = 0; i < 1; i++) {
        test_kernel<<<1,5>>>(cudaTest, d_time);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        gpuErrchk( memcpyDeviceToHost(&test.v, &cudaTest.v) );
        gpuErrchk( memcpyDeviceToHost(&test.m, &cudaTest.m) );
        gpuErrchk( hipMemcpy(&time, d_time, sizeof(unsigned long long), hipMemcpyDeviceToHost) );
        cout << time << endl;
        cout << test.v << endl;
        cout << test.m << endl;
    }
}
