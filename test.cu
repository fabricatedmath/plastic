#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <Eigen/Dense>
#include "test.h"
#include "err.cuh"
#include "randGen.cuh"
#include "constants.h"
#include "cuda_state.cuh"
#include "input.cuh"
#include "hip/hip_runtime_api.h"
#include "cuda_utility.cuh"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>

using namespace std;
using namespace Eigen;
namespace cg = cooperative_groups;

const int rows = 50;
const int numThreads = 128;

typedef RandomGen<hiprandState> Rgen;

__global__ void spin_kernel() {
    unsigned long long startTime = clock64();
    unsigned long long hz = 2100000000;
    unsigned long long seconds = 100;
    unsigned long long thresh = hz * seconds;
    do {
    } while ((clock64() - startTime) < thresh);
}

__device__ void fillBuffers(const CudaMatrixXf input, CudaMatrixXf lgnfirings, CudaMatrixXi poissonNoise, Rgen rgen, int inputRow) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    const float* rowPtr = getRowPtr(input, inputRow);
    hiprandState g = rgen.get(id);
    float* lgnfiringsRowPtr;
    const unsigned int tid = threadIdx.x;
    for (int row = blockIdx.x; row < NBSTEPSSTIM; row += gridDim.x) {
        lgnfiringsRowPtr = getRowPtr(lgnfirings, row);
        for (int i = tid; i < FFRFSIZE; i += blockDim.x) {
            float rand = rgen.sampleUniform(tid,&g);
            lgnfiringsRowPtr[i] = rand < rowPtr[i];
        }
    }

    int* poissonNoiseRowPtr; 
    for (int row = blockIdx.x; row < NBSTEPSPERPRES; row += gridDim.x) {
        poissonNoiseRowPtr = getRowPtr(poissonNoise, row);
        for (int i = tid; i < NBNEUR; i += blockDim.x) {
            int rand1 = rgen.samplePosPoisson(tid,&g);
            int rand2 = rgen.sampleNegPoisson(tid,&g);
            poissonNoiseRowPtr[i] = rand1 + rand2;
        }
    }
    rgen.put(id,g);
}

__global__ void test_kernel(CudaMutableState ms,
                            const CudaStaticState ss,
                            CudaBuffers b,
                            Rgen rgen,
                            unsigned long long* time) {
    
    unsigned long long startTime = clock64();
    __shared__ float sdata[numThreads];
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(block);

    cg::grid_group grid = cg::this_grid();

    const unsigned int tid = block.thread_rank();
    
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    for (int inputRow = 0; inputRow < 100; inputRow++) {
        fillBuffers(ss.input, b.lgnfirings, b.poissonNoise, rgen, inputRow);
//        fillBuffers(ss.input, b.lgnfirings, rgen, inputRow);
        cg::sync(grid);
        for (int numStepsThisPres = 0; numStepsThisPres < NBSTEPSPERPRES; numStepsThisPres++) {
            /* Calculate Inputs with block per Neuron */
            for(int row = blockIdx.x; row < NBNEUR; row += gridDim.x) {
                float iff = 0;
                if (numStepsThisPres < NBSTEPSSTIM) {
                    iff = VSTIM * computeIFFNeuron(sdata, block, tile32, tid, ms.wff, b.lgnfirings, numStepsThisPres, row);
                }

                float ilat = LATCONNMULT * VSTIM * computeILATNeuron(sdata, block, tile32, tid, ms.w, ms.incomingSpikes, ms.firings, ss.delays, row);

                int* noiseRowPtr = getRowPtr(b.poissonNoise, numStepsThisPres);
                float noise = noiseRowPtr[row];
                b.neuronInputs.data[row] = iff + ilat + noise;
            }
            /* Sync blocks from Input calculation */
            cg::sync(grid);

            /* Neuron per thread stuff */
            for (int neuron = id; neuron < NBNEUR; neuron += gridDim.x) {
                float v = ms.v.data[id];
                float vprev = v;
                float vthresh = ms.vthresh.data[id];
                float input = b.neuronInputs.data[id];
                float wadap = ms.wadap.data[id];
                float z = ms.z.data[id];
                int isSpiking = ms.isSpiking.data[id];
                int firing = ms.firings.data[id];
                float vlongtrace = ms.vlongtrace.data[id];
                float xplastLat = ms.xplastLat.data[id];
                float xplastFF = ms.xplastFF.data[id];
                
                float lgnfirings = 0;
                if (numStepsThisPres < NBSTEPSSTIM) {
                    float* rowLgnFirings = getRowPtr(b.lgnfirings, numStepsThisPres);
                    lgnfirings = rowLgnFirings[id];
                }

                float vneg = ms.vneg.data[id];
                float vpos = ms.vpos.data[id];

                /* PRE-SPIKE UPDATE */
                v += (DT/CONSTC) * (-GLEAK * (v - ELEAK) + GLEAK * DELTAT * expf((v-vthresh) / DELTAT) + z - wadap) + input;

                if (isSpiking > 1) {
                    v = VPEAK-0.001;
                }

                if (isSpiking == 1) {
                    v = VRESET;
                    z = ISP;
                    vthresh = VTMAX;
                    wadap += CONSTB;
                }
                isSpiking = max(0,isSpiking - 1);
                v = max(v,MINV);

                /* SPIKE UPDATE */
                firing = 0;
                if (v > VPEAK) {
                    firing = 1;
                    v = VPEAK;
                    isSpiking = NBSPIKINGSTEPS;
                }

                /* POST-SPIKE UPDATE */
                wadap = wadap + (DT / TAUADAP) * (CONSTA * (v - ELEAK) - wadap);
                z = z + (DT / TAUZ) * (-1.0) * z;
                vthresh = vthresh + (DT / TAUVTHRESH) * (-1.0 * vthresh + VTREST);
                vlongtrace = vlongtrace + (DT / TAUVLONGTRACE) * (max(0.0,(vprev - THETAVLONGTRACE)) - vlongtrace);

                xplastLat = xplastLat + firing / TAUXPLAST - (DT / TAUXPLAST) * xplastLat;
                xplastFF = xplastFF + lgnfirings / TAUXPLAST - (DT / TAUXPLAST) * xplastFF;

                float altds = ss.altds.data[id];

                /* PLASTICITY */
                
                b.eachNeurLTD.data[id] = DT * (-altds / VREF2) * vlongtrace * vlongtrace * max(0.0,vneg - THETAVNEG);
                b.eachNeurLTP.data[id] = DT * ALTP * ALTPMULT * max(0.0, vpos - THETAVNEG) * max(0.0, v - THETAVPOS);

                
                ms.v.data[id] = v;
            }

            /* Plasticity */
        }
    }
    unsigned long long endTime = clock64();
    if (id == 0) {
        *time = (endTime - startTime);
    }   
}

void printSetBlockGridStats(int* thisNumBlocks, int* thisNumThreads) {
    const char *argv[] = {""};
    
    int argc = 0;
    int device = findCudaDevice(argc, argv);
    hipDeviceProp_t prop = { 0 };
    //    cout << gpuGetMaxGflopsDeviceId() << endl;
    gpuErrchk( hipSetDevice(device) );
    gpuErrchk( hipGetDeviceProperties(&prop, device) );

//    int numThreads = 128;
    int maxBlocks = prop.multiProcessorCount * (prop.maxThreadsPerMultiProcessor / prop.maxThreadsPerBlock);
    int numSms = prop.multiProcessorCount;
    int numBlocksPerSm = 0;
    checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, test_kernel, numThreads, 0));
    int numBlocks = numBlocksPerSm * numSms;
    
    cout << "--------Grid/Block Statistics-----------------------------------------" << endl;
    cout << endl;
    cout << "\t\tGlobally" << endl;
    printf("\tSMs:\t\t\t%d\n", numSms);
    printf("\tMax Blocks:\t\t%d\n", maxBlocks);
    cout << endl;
    cout << "\t\tProgram" << endl;
    printf("\tBlocks per SM:\t\t%d\n", numBlocksPerSm);
    printf("\tBlocks:\t\t\t%d\n", numBlocks);
    printf("\tThreads per Block:\t%d\n", numThreads);
    printf("\tThreads:\t\t%d\n", numThreads * numBlocks);
    cout << endl;
    cout << "----------------------------------------------------------------------" << endl;

    *thisNumBlocks = numBlocks;
    *thisNumThreads = numThreads;
}

void wrapper(MutableState mutableState, StaticState staticState, Buffers buffers) {
    int numBlocks;
    int numThreads;
    
    printSetBlockGridStats(&numBlocks,&numThreads);

    size_t usedBeforeAllocation; 
    {
        size_t freeBytes;
        size_t totalBytes;
        size_t usedBytes;
        gpuErrchk( hipMemGetInfo(&freeBytes,&totalBytes) );
        usedBytes = totalBytes - freeBytes;
        usedBeforeAllocation = usedBytes;
        cout << "--------Memory Pre-Allocation-----------------------------------------" << endl;
        cout << endl;
        printf("\tfree:\t\t%d MB\n", freeBytes / (1024*1024));
        printf("\tused:\t\t%d MB\n", usedBytes / (1024*1024));
        printf("\ttotal:\t\t%d MB\n", totalBytes / (1024*1024));
        cout << endl;
        cout << "----------------------------------------------------------------------" << endl;
    }

    unsigned long long time;
    unsigned long long* d_time;
    gpuErrchk( hipMalloc(&d_time, sizeof(unsigned long long)) );
    
    CudaMutableState cudaMutableState;
    gpuErrchk( hipMalloc(&mutableState,&cudaMutableState) );
    gpuErrchk( memcpyHostToDevice(&mutableState,&cudaMutableState) );

    CudaStaticState cudaStaticState;
    gpuErrchk( hipMalloc(&staticState,&cudaStaticState) );
    gpuErrchk( memcpyHostToDevice(&staticState,&cudaStaticState) );

    CudaBuffers cudaBuffers;
    gpuErrchk( hipMalloc(&buffers,&cudaBuffers) );
    gpuErrchk( memcpyHostToDevice(&buffers,&cudaBuffers) );

    numBlocks = min(NBNEUR,numBlocks);
    cout << "num blocks: " << numBlocks << endl;
    typedef RandomGen<hiprandState> Rgen;
    Rgen cudaRgen(numBlocks, numThreads, 1.1, 1.8);

    size_t usedAfterAllocation;
    size_t allocated;
    {
        size_t freeBytes;
        size_t totalBytes;
        size_t usedBytes;
        gpuErrchk( hipMemGetInfo(&freeBytes,&totalBytes) );
        usedBytes = totalBytes - freeBytes;
        usedAfterAllocation = usedBytes;
        allocated = usedAfterAllocation - usedBeforeAllocation;
        cout << "--------Memory Post-Allocation----------------------------------------" << endl;
        cout << endl;
        printf("\tfree:\t\t%d MB\n", freeBytes / (1024*1024));
        printf("\tused:\t\t%d MB\n", usedBytes / (1024*1024));
        printf("\ttotal:\t\t%d MB\n", totalBytes / (1024*1024));
        cout << endl;
        printf("\tallocated:\t%d MB\n", allocated / (1024*1024));
        cout << endl;
        cout << "----------------------------------------------------------------------" << endl;
    }

    void *kernelArgs[] = {
        (void*)&cudaMutableState,
        (void*)&cudaStaticState,
        (void*)&cudaBuffers,
        (void*)&cudaRgen,
        (void*)&d_time
    };

    void *spinKernelArgs[] = {
    };

    const dim3 dimBlock(numThreads,1,1);
    const dim3 dimGrid(numBlocks,1,1);
    
    const int smemSize = 0;
    for (int i = 0; i < 10; i++) {
        gpuErrchk( hipLaunchCooperativeKernel((void*)test_kernel, dimGrid, dimBlock, kernelArgs, smemSize, NULL) );
        //gpuErrchk( hipLaunchCooperativeKernel((void*)spin_kernel, dimGrid, dimBlock, spinKernelArgs, smemSize, NULL) );
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        gpuErrchk( memcpyDeviceToHost(&mutableState, &cudaMutableState) );
        gpuErrchk( memcpyDeviceToHost(&buffers, &cudaBuffers) );
        gpuErrchk( hipMemcpy(&time, d_time, sizeof(unsigned long long), hipMemcpyDeviceToHost) );
        cout << time << endl;
/*
        for (int row = 0; row < buffers.lgnfirings.rows(); row++) {
            cout << buffers.lgnfirings.row(row) << endl;
        }
        */
    }
}


