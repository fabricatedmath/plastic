#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <Eigen/Dense>
#include "test.h"
#include "err.cuh"
#include "mem.cuh"

using namespace std;
using namespace Eigen;

__global__ void test_kernel(CudaTest test, unsigned long long* time) {
    unsigned long long startTime = clock();
    float v = test.v.data[threadIdx.x];
    test.v.data[threadIdx.x] = v+1;
    for (int row = 0; row < 5; row++) {
        float* rowPtr = getRowPtr(test.m, row);
        float i = rowPtr[threadIdx.x];
        rowPtr[threadIdx.x] = i+1;
    }
    unsigned long long endTime = clock();
    *time = (endTime - startTime);
}

void wrapper(Test test) {
    unsigned long long time;
    unsigned long long* d_time;
    gpuErrchk( hipMalloc(&d_time, sizeof(unsigned long long)) );
    CudaTest cudaTest;
    gpuErrchk( hipMalloc(&test,&cudaTest) );
    gpuErrchk( memcpyHostToDevice(&test,&cudaTest) );

    for (int i = 0; i < 10; i++) {
        test_kernel<<<1,5>>>(cudaTest, d_time);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        gpuErrchk( memcpyDeviceToHost(&test, &cudaTest) );
        gpuErrchk( hipMemcpy(&time, d_time, sizeof(unsigned long long), hipMemcpyDeviceToHost) );
        cout << time << endl;
        cout << test.v << endl;
        cout << test.m << endl;
    }
}
