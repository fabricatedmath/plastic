#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <Eigen/Dense>
#include "test.h"

using namespace std;
using namespace Eigen;

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    cout << x << endl; \
    exit(-1); \
    }} while(0)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct CudaMatrixXf {
    float* data;
    size_t pitch;
};

struct CudaVectorXf {
    float* data;
};

struct CudaTest {
    CudaMatrixXf m;
    CudaVectorXf v;
};

__global__ void test_kernel(CudaTest test, unsigned long long* time) {
    unsigned long long startTime = clock();
    float v = test.v.data[threadIdx.x];
    printf("dogs2:%f\n",v);
    test.v.data[threadIdx.x] = v+1;
    unsigned long long endTime = clock();
    *time = (endTime - startTime);
}

__global__ void test_kernel2(unsigned long long* time) {
    unsigned long long startTime = clock();
    printf("dogs2\n");
    unsigned long long endTime = clock();
    *time = (endTime - startTime);
}
/*
__global__ void test_kernel2(flotest) {
    float v = test.v.data[0];
    printf("dogs2\n%d",v);
    }*/

hipError_t hipMalloc(VectorXf* v, CudaVectorXf* cv) {
    cout << v->size() << endl;
    return hipMalloc((void**)&cv->data, v->size() * sizeof(float));
}

hipError_t memcpyHostToDevice(VectorXf* v, CudaVectorXf* cv) {
    return hipMemcpy((void**)cv->data, v->data(), v->size() * sizeof(float), hipMemcpyHostToDevice);
}

hipError_t memcpyDeviceToHost(VectorXf* v, CudaVectorXf* cv) {
    return hipMemcpy((void**)v->data(), cv->data, v->size() * sizeof(float), hipMemcpyDeviceToHost);
}

void hipMalloc(MatrixXf m, CudaMatrixXf cm) {
    hipMallocPitch((void**)&cm.data, &cm.pitch, m.cols() * sizeof(float), m.rows());
}

void wrapper(Test test) {
    cout << "cats" << endl;
    //cout << test.v << endl;
    unsigned long long time;
    unsigned long long* d_time;
    gpuErrchk( hipMalloc(&d_time, sizeof(unsigned long long)) );
    CudaTest cudaTest;
    CudaVectorXf cv;
    gpuErrchk( hipMalloc(&test.v, &cudaTest.v) );
    gpuErrchk( memcpyHostToDevice(&test.v, &cudaTest.v) );
    //gpuErrchk( hipMalloc((void**)&cv.data, 5 * sizeof(float)) );
    test_kernel<<<1,5>>>(cudaTest, d_time);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( memcpyDeviceToHost(&test.v, &cudaTest.v) );
    hipMemcpy(&time, d_time, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    cout << time << endl;
    cout << test.v << endl;
    
//    float* data = (float*)malloc(5*sizeof(float));
//    hipMemcpy(data, cudaTest.v.data, 5 * sizeof(float), hipMemcpyDeviceToHost);
//    cout << data[0] << endl;
}
