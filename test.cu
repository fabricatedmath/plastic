#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <Eigen/Dense>
#include "test.h"
#include "err.cuh"
#include "randGen.cuh"
#include "cuda_state.cuh"
#include "hip/hip_runtime_api.h"
#include "cuda_utility.cuh"

using namespace std;
using namespace Eigen;

const int rows = 50;

__global__ void test_kernel(CudaMutableState cudaMutableState, unsigned long long* time) {
    unsigned long long startTime = clock();
//    float w = cudaMutableState.w.data[threadIdx.x];
//    cudaMutableState.w.data[threadIdx.x] = w+1;
    for (int row = 0; row < rows; row++) {
        float* rowPtr = getRowPtr(cudaMutableState.w, row);
        float i = rowPtr[threadIdx.x];
        rowPtr[threadIdx.x] = i+1;
        printf("%d",threadIdx.x);
    }
    unsigned long long endTime = clock();
    *time = (endTime - startTime);
}

void wrapper2() {
    typedef RandomGen<hiprandState,1,2> Rgen;
    Rgen d_rgen(1.1,1.8);
    //Rgen* d_rgen = new Rgen(1.8,1.1);
}

void something() {
    char array[] = "dogs";
    char *p = array;
    const char *arg = p;
    const char *argv[] = {"dogs"};
    
    int argc = 0;
    int device = findCudaDevice(argc, argv);
    hipDeviceProp_t prop = { 0 };
    cout << gpuGetMaxGflopsDeviceId() << endl;
    gpuErrchk( hipSetDevice(device) );
    gpuErrchk( hipGetDeviceProperties(&prop, device) );
    cout << prop.multiProcessorCount << endl;
    cout << prop.maxThreadsPerMultiProcessor << endl;
    cout << prop.maxThreadsPerBlock << endl;
    cout << endl;

    int numThreads = 120; //prop.maxThreadsPerMultiProcessor;
    int maxBlocks = prop.multiProcessorCount * (prop.maxThreadsPerMultiProcessor / prop.maxThreadsPerBlock);
    int numSms = prop.multiProcessorCount;
    int numBlocksPerSm = 0;
    checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, test_kernel, numThreads, 0));
    cout << numBlocksPerSm << endl;
    cout << numBlocksPerSm * numSms << endl;
}

void wrapper(MutableState mutableState, StaticState staticState) {
    something();
//    gpuErrchk( hipOccupancyMaxActiveBlocksPerMultiprocessor( );
    unsigned long long time;
    unsigned long long* d_time;
    gpuErrchk( hipMalloc(&d_time, sizeof(unsigned long long)) );
    CudaMutableState cudaMutableState;
    gpuErrchk( hipMalloc(&mutableState,&cudaMutableState) );
    gpuErrchk( memcpyHostToDevice(&mutableState,&cudaMutableState) );


    void *kernelArgs[] = {
        (void*)&cudaMutableState,
        (void*)&d_time
    };

    int numBlocks = 544;
    int numThreads = 120;

    dim3 dimBlock(numThreads,1,1);
    dim3 dimGrid(numBlocks,1,1);

    const int smemSize = 0;
    for (int i = 0; i < 10; i++) {
        gpuErrchk( hipLaunchCooperativeKernel((void*)test_kernel, dimGrid, dimBlock, kernelArgs, smemSize, NULL) );
//        test_kernel<<<1,5>>>(cudaMutableState, d_time);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        gpuErrchk( memcpyDeviceToHost(&mutableState, &cudaMutableState) );
        gpuErrchk( hipMemcpy(&time, d_time, sizeof(unsigned long long), hipMemcpyDeviceToHost) );
        cout << time << endl;
    }
}


