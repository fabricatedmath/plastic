#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_cooperative_groups.h>

#include "eigen.h"
#include "err.cuh"
#include "cuda_eigen.cuh"

namespace cg = cooperative_groups;
using namespace std;

struct State {
    VectorX<float> f;
};

struct CudaState {
    CudaVectorX<float> f;
};

hipError_t hipMalloc(State* s, CudaState* cs) {
    errRet( hipMalloc(&s->f,&cs->f) );
    return hipSuccess;
}

hipError_t memcpyHostToDevice(State* s, CudaState* cs) {
    errRet( memcpyHostToDevice(&s->f,&cs->f) );
    return hipSuccess;
}

hipError_t memcpyDeviceToHost(State* s, CudaState* cs) {
    errRet( memcpyDeviceToHost(&s->f,&cs->f) );
    return hipSuccess;
}

__global__ void test(int* const __restrict__ d1,
                     int* const __restrict__ d2) {
    cg::multi_grid_group myMultiGrid = cg::this_multi_grid();
    cg::grid_group myGrid = cg::this_grid();
    cg::thread_block myBlock = cg::this_thread_block();

    if (myGrid.thread_rank() == 0) {
        printf("%d\n", myMultiGrid.grid_rank());
    }

    cg::sync(myMultiGrid);

    if (myMultiGrid.thread_rank() == 0) {
        printf("%d : %d \n", *d1, *d2);
    }

    cg::sync(myMultiGrid);

    if(myMultiGrid.grid_rank() == 0) {
        *d1 = 1;
    }

    if(myMultiGrid.grid_rank() == 1) {
        *d2 = 1;
    }
    
    cg::sync(myMultiGrid);

    if (myMultiGrid.thread_rank() == 0) {
        printf("%d : %d \n", *d1, *d2);
    }
}

const int numDevices = 2;

void call() {

    std::vector<int*> thisD(numDevices);

    gpuErrchk( hipSetDevice(0) );
    gpuErrchk( hipDeviceEnablePeerAccess(1,0) );

    gpuErrchk( hipSetDevice(1) );
    gpuErrchk( hipDeviceEnablePeerAccess(0,0) );

    for (int i = 0; i < numDevices; i++) {
        gpuErrchk( hipSetDevice(i) );
        gpuErrchk( hipMalloc((void **)&thisD[i], numDevices*sizeof(int)) );
        gpuErrchk( hipMemset(thisD[i], 0, numDevices*sizeof(int)) );
    }

    const dim3 dimBlock(32,1,1);
    const dim3 dimGrid(1,1,1);
    const int smemSize = 0;
    hipLaunchParams launchParamsList[numDevices];
    for (int i = 0; i < numDevices; i++) {
        gpuErrchk( hipSetDevice(i) );
        launchParamsList[i].func      = (void *)test;
        launchParamsList[i].blockDim  = dimBlock;
        launchParamsList[i].gridDim   = dimGrid;
        launchParamsList[i].args      = (void **)malloc(2 * sizeof(void *));
        {
            launchParamsList[i].args[0] = &thisD[0];
            launchParamsList[i].args[1] = &thisD[1];
        }
        launchParamsList[i].sharedMem = smemSize;

        gpuErrchk( hipStreamCreate(&launchParamsList[i].stream) );
    }

    gpuErrchk( hipLaunchCooperativeKernelMultiDevice(launchParamsList, 2, hipCooperativeLaunchMultiDeviceNoPreSync |
          hipCooperativeLaunchMultiDeviceNoPostSync) );
    
    for (int i = 0; i < numDevices; i++) {
        gpuErrchk( hipSetDevice(i) );
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
    }
}

