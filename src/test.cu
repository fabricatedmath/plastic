#include "hip/hip_runtime.h"
#include <iostream>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <Eigen/Dense>
#include "test.h"
#include "err.cuh"
#include "randGen.cuh"
#include "constants.h"
#include "cuda_state.cuh"
#include "hip/hip_runtime_api.h"
#include "kernels/robust_kernel/kernel.cuh"
#include "stats.cuh"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>

using namespace std;
using namespace Eigen;
namespace cg = cooperative_groups;

const int numThreadsConstant = 128;

void wrapper(MutableState mutableState, StaticState staticState, Buffers buffers) {
    int numThreads = numThreadsConstant;

    auto func = test_kernel<numThreadsConstant>;
    
    int numBlocks = printSetBlockGridStats(func, numThreadsConstant);

    unsigned long long time;
    unsigned long long* d_time;
    gpuErrchk( hipMalloc(&d_time, sizeof(unsigned long long)) );
    
    CudaMutableState cudaMutableState;
    gpuErrchk( hipMalloc(&mutableState,&cudaMutableState) );
    gpuErrchk( memcpyHostToDevice(&mutableState,&cudaMutableState) );

    CudaStaticState cudaStaticState;
    gpuErrchk( hipMalloc(&staticState,&cudaStaticState) );
    gpuErrchk( memcpyHostToDevice(&staticState,&cudaStaticState) );

    CudaBuffers cudaBuffers;
    gpuErrchk( hipMalloc(&buffers,&cudaBuffers) );
    gpuErrchk( memcpyHostToDevice(&buffers,&cudaBuffers) );

    printNvidiaSmi();

    numBlocks = min(NBNEUR,numBlocks);
    cout << "Num Blocks: " << numBlocks << endl;
    
    typedef RandomGen<hiprandState> Rgen;
    Rgen cudaRgen(numBlocks, numThreads, 1.1, 1.8);

    void *kernelArgs[] = {
        (void*)&cudaMutableState,
        (void*)&cudaStaticState,
        (void*)&cudaBuffers,
        (void*)&cudaRgen,
        (void*)&d_time
    };

    const dim3 dimBlock(numThreads,1,1);
    const dim3 dimGrid(numBlocks,1,1);
    const int smemSize = 0;
    
    for (int i = 0; i < 10; i++) {
        gpuErrchk( hipLaunchCooperativeKernel((void*)func,  dimGrid, dimBlock, kernelArgs, smemSize, NULL) );
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        gpuErrchk( memcpyDeviceToHost(&mutableState, &cudaMutableState) );
        gpuErrchk( memcpyDeviceToHost(&buffers, &cudaBuffers) );
        gpuErrchk( hipMemcpy(&time, d_time, sizeof(unsigned long long), hipMemcpyDeviceToHost) );
        cout << "Clocks: " << time << endl;
    }
}


